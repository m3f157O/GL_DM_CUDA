#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////

// Pointers for converted vector

int* x_array;
int* y_array;
int* dangling_array;
double *pr_array, *val_array, *pr_tmp_array;
// Pointers for VRAM data
int *x_gpu, *y_gpu;
double *val_gpu, *pr_gpu;

// Temporary arrays
double *dangling_factor_gpu;

double *pr_tmp_gpu;
int *dangling_gpu;
int V_size;
int E_size;
int dangling_size;


// Write GPU kernel here!

//////////////////////////////
//////////////////////////////
__global__ void cuda_hello(){
    printf("this is working");

}
/*
 * inline void spmv_coo_cpu(const int *x, const int *y, const double *val, const double *vec, double *result, int N) {
    for (int i = 0; i < N; i++) {
        //// the value of each node is the summation of the value of outgoing nodes * the previous pagerank score
        result[x[i]] += val[i] * vec[y[i]];
    }
}
 */
__global__ void spmv_coo_gpu(const int *x_gpu, const int *y_gpu, const double *val_gpu, const double *pr_gpu, double *pr_tmp_gpu, int V){


    int i = threadIdx.x + blockIdx.x * blockDim.x;
    atomicAdd(&pr_tmp_gpu[x_gpu[i]], val_gpu[i] * pr_gpu[y_gpu[i]]);

}

__global__ void dot_product_gpu(int *dangling_bitmap_gpu, double *pr_gpu, int V, double *dangling_factor_gpu) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    //dangling_factor += dangling_bitmap[i] * pr[i];

    /*printf("THIS IS Dangling: ");
    printf("%f\n",dangling_bitmap_gpu[i]);*/

    atomicAdd(dangling_factor_gpu, dangling_bitmap_gpu[i] * pr_gpu[i]);
    //printf("%f",dangling_factor_gpu[0]);
}

__global__ void initKernel(double *pr_gpu, int len)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<len)
    {
        pr_gpu[i] = 1.0/len;
        printf("%f",pr_gpu[i]);
        //printf("%f",pr_gpu[i]);
    }
}


__global__ void axbp_custom(double alpha, double one_minus_a, double* pr_tmp, double alpha_dangling_onV, int personalization_vertex, double* pr_tmp_result, int len)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<len)
    {
        //if(y_gpu[i])
        pr_tmp_result[i]=alpha * pr_tmp[i] + alpha_dangling_onV + ((personalization_vertex == i) ? one_minus_a : 0.0);
        //printf("%f\n",devPtr[i]);
        printf("pr temp gpu : %f\n",pr_tmp_result[i]);

    }


}




__global__ void initKernelInverseValue(double *devPtr,const int *y_gpu, int* outdegree_gpu ,const int len)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i<len)
    {
        //if(y_gpu[i])
        devPtr[i] = 1.0/outdegree_gpu[y_gpu[i]];
        //printf("%f\n",devPtr[i]);
    }
    //printf("%d\n",i);


}



// CPU Utility inversfunctions;
void personalized_pagerank_gpu_support(
        const int V,
        const int E,
        const int *dangling_bitmap,
        const int personalization_vertex,
        double *pr,
        double *val,
        double alpha=DEFAULT_ALPHA,
        double convergence_threshold=DEFAULT_CONVERGENCE,
        const int max_iterations=DEFAULT_MAX_ITER
        ){

    // Temporary PPR result;

    int iter = 0;  //stay on cpu
    bool converged = false; //stay on cpu
    while (!converged && iter < max_iterations) {     //stay on cpu


        hipMemset(pr_tmp_gpu, 0, V_size);  // ???

        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
            printf("from pr memset Error: %s\n", hipGetErrorString(err));


        err = hipGetLastError();
        if (err != hipSuccess)
            printf("from dangling memset Error: %s\n", hipGetErrorString(err));

        int N = E;
        int threads_per_block = std::min(1,V);   //todo fix this
        int num_blocks = N / threads_per_block;
        // Launch add() kernel on GPU

        //spmv_coo_gpu<<<num_blocks,threads_per_block>>>(x_gpu, y_gpu, val_gpu, pr_gpu, pr_tmp_gpu,V);
        spmv_coo_gpu<<<num_blocks,threads_per_block>>>(x_gpu, y_gpu, val_gpu, pr_gpu, pr_tmp_gpu,V);

        err = hipGetLastError();
        if (err != hipSuccess)
            printf("from spmv  Error: %s\n", hipGetErrorString(err));

        hipDeviceSynchronize();


        dot_product_gpu<<<num_blocks,threads_per_block>>>(dangling_gpu, pr_gpu, V, dangling_factor_gpu);

        err = hipGetLastError();
        if (err != hipSuccess)
            printf("from dot product gpu  Error: %s\n", hipGetErrorString(err));


        double dangling;
        err = hipMemcpy(&dangling, dangling_gpu, sizeof(double) ,hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr,
                    "Failed to copy dangling from device to host (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }


        double new_dangling=dangling*alpha/V;
        std::cout << dangling << "\n";


        double one_minus_a=1-alpha;
        axbp_custom<<<1,1>>>(alpha,one_minus_a, pr_tmp_gpu,new_dangling,personalization_vertex,pr_tmp_gpu,V);


        ////alpha choose next link, pr_tmp, beta is a priori probability (?????) that next chosen is dangling over V
        //axpb_personalized_cpu(alpha, pr_tmp, alpha * dangling_factor / V, personalization_vertex, pr_tmp, V); //TODO GPU

        // Check convergence;
        //double err = euclidean_distance_cpu(pr, pr_tmp, V); //TODO GPU
        //converged = err <= convergence_threshold; // ????*/

        err = hipMemcpy(val, val_gpu, V_size ,hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr,
                    "Failed to copy IN PAGERANK GPU SUPPORT from device to host (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
     /**   std::ostringstream out;
        out.precision(3);
        std::cout << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            std::cout << val[i] << ", ";
        }
        std::cout << "...]";*/

        // Update the PageRank vector;

      /*  err = hipMemcpy(pr_gpu, pr_tmp_gpu, V_size ,hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
            fprintf(stderr,
                    "Failed to copy IN PAGERANK GPU SUPPORT from device to host (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }*/
        err = hipMemcpy(pr_array, pr_gpu, V_size ,hipMemcpyDeviceToHost);
        /*if (err != hipSuccess) {
            fprintf(stderr,
                    "Failed to copy IN PAGERANK GPU SUPPORT from device to host (error code %s)!\n",
                    hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }*/
        iter++;
    }




    //free(pr_tmp);
}
// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        true                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;   /////this is done in initKernelOutdegree
        //printf("%d",outdegree[y[i]]);
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
    //// each node val is dependent on the number of outgoing edges
        val[i] = 1.0 / outdegree[y[i]];  /////this is done in initKernelInverse
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc() {
    // Load the input graph and preprocess it;
    initialize_graph();

    // Allocate any GPU data here;
    // TODO!

    // Size of allocations
    V_size = V * sizeof(double);
    dangling_size = V * sizeof(int);
    E_size = E * sizeof(int);



    // Allocate space in VRAM
    hipError_t err = hipSuccess;
    err = hipMalloc((void **)&x_gpu, E_size);
    hipMalloc((void **)&y_gpu, E_size);
    hipMalloc((void **)&val_gpu, V_size);
    hipMalloc((void **)&pr_gpu, V_size);
    hipMalloc((void **)&pr_tmp_gpu, V_size);
    hipMalloc((void **)&dangling_gpu, dangling_size);
    hipMalloc((void **)&dangling_factor_gpu, sizeof(double));

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vectors (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Initialize data;
void PersonalizedPageRank::init() {
    // Do any additional CPU or GPU setup here;
    // TODO!

    std::cout << "INITIALIZING";
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset() {
    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr.begin(), pr.end(), 1.0 / V);
    // Generate a new personalization vertex for this iteration;
    personalization_vertex = rand() % V;
    if (debug) std::cout << "personalization vertex=" << personalization_vertex << std::endl;
    std::cout << "RESET";

    x_array = &x[0];
    y_array = &y[0];
    dangling_array = &dangling[0];

    pr_array = &pr[0];
    val_array = &val[0];
    std::vector<double> pr_tmp;
    pr_tmp_array = &pr_tmp[0];
    /*for(int i=0;i<V;i++)
    {
        std::cout << dangling_array[i] ;
        std::cout << "THIS IS THE DOG\n";

    }*/




    //todo val is missing
    // Do any GPU reset here, and also transfer data to the GPU;
    // TODO!
    hipError_t err = hipSuccess;
    hipMemcpy(x_gpu, x_array, E_size, hipMemcpyHostToDevice);
    err = hipMemcpy(y_gpu, y_array, E_size, hipMemcpyHostToDevice);
    hipMemcpy(pr_gpu, pr_array, V_size, hipMemcpyHostToDevice);
    hipMemcpy(pr_tmp_gpu, pr_tmp_array, V_size, hipMemcpyHostToDevice);
    hipMemcpy(val_gpu, val_array, V_size, hipMemcpyHostToDevice);
    hipMemcpy(dangling_gpu, dangling_array, dangling_size, hipMemcpyHostToDevice);

    printf("%d",personalization_vertex);
    if (err != hipSuccess) {
        fprintf(stderr,
                "Failed to copy IN RESET from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from RAM to VRAM




}

void PersonalizedPageRank::execute(int iter) {
    // Do the GPU computation here, and also transfer results to the CPU;
    //TODO! (and save the GPU PPR values into the "pr" array)
    int N = V;
    int threads_per_block = std::min(1,V);   //todo make sure that num blocks is always >0
    int num_blocks = N / threads_per_block;

    initKernel<<<num_blocks,threads_per_block>>> ( pr_gpu , V );





    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("init kernel Error: %s\n", hipGetErrorString(err));
    std::cout << "EXECUTE";

    personalized_pagerank_gpu_support(V, E, dangling.data(), personalization_vertex, pr.data(), val.data(), alpha, 1e-6, 100);



}

void PersonalizedPageRank::cpu_validation(int iter) {

    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);

    ////// INITIALIZE DUMMY VECTOR WITH 1/V FLOAT
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    /////// pass x (starting edge vertex) y (arriving edge vertex)  OK
    /// val ( is all one at starting )
    /// pr_golden (pagerank scores)
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, 1e-6, 100);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::unordered_set<int> top_pr_indices;
    std::unordered_set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++) {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug) {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu) {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            } else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > 1e-6) {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug) std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(precision);
    } else {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}


void PersonalizedPageRank::clean() {
    // Delete any GPU data or additional CPU data;
    // TODO!
    hipFree(x_gpu);
    hipFree(y_gpu);
    hipFree(val_gpu);
    hipFree(pr_gpu);
    hipFree(pr_tmp_gpu);
}
