#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "personalized_pagerank.cuh"
#include "hipblas.h"

#define DEFAULT_THREADS_PER_BLOCK_VERTEX 64
#define DEFAULT_THREADS_PER_BLOCK 1024

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////

// Pointers for converted vector




// Write GPU kernel here!

//////////////////////////////
//////////////////////////////

__global__ void spmv_coo_gpu(const int *x_gpu, const int *y_gpu, const double *val_gpu, const double *pr_gpu, double *pr_tmp_gpu, int *E, int *V) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i=thread_id;i<(*V);i+=blockDim.x*gridDim.x){
        pr_tmp_gpu[i] = 0;
    }
    __syncthreads();
    for(int i=thread_id;i<(*E);i+=blockDim.x*gridDim.x) {
        atomicAdd(&pr_tmp_gpu[x_gpu[i]], val_gpu[i] * pr_gpu[y_gpu[i]]);
    }
}

__global__ void spmv_coo_gpu_2(const int *x_gpu, const int *y_gpu, const double *val_gpu, const double *pr_gpu, double *pr_tmp_gpu, int *E, int *V) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i=thread_id;i<(*V);i+=blockDim.x*gridDim.x){
        pr_tmp_gpu[i] = 0;
    }
    __syncthreads();
    int tid = threadIdx.x;
    int i = tid+blockIdx.x*blockDim.x;
    if(i<(*E)) {
        __shared__ double temp[DEFAULT_THREADS_PER_BLOCK];
        __shared__ int idx[DEFAULT_THREADS_PER_BLOCK];
        temp[tid] = val_gpu[i] * pr_gpu[y_gpu[i]];
        idx[tid] = x_gpu[i];
        __syncthreads();
        atomicAdd(&pr_tmp_gpu[idx[tid]], temp[tid]);
    }
}

__global__ void dot_product_gpu(int *dangling_gpu, double *pr_gpu, int *V, double *dangling_factor_gpu) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id == 0) (*dangling_factor_gpu)=0;
    for(int i=thread_id;i<(*V);i+=blockDim.x*gridDim.x){
        atomicAdd(dangling_factor_gpu, dangling_gpu[i] * pr_gpu[i]);
    }
}

__global__ void dot_product_gpu_2(int *dangling, double *pr, int *V, double *dangling_factor) {

    int tid = threadIdx.x;
    int i = tid+blockIdx.x*blockDim.x;

    if(i == 0) (*dangling_factor)=0;

    if(i<(*V)) {
        __shared__ double temp[DEFAULT_THREADS_PER_BLOCK_VERTEX];
        temp[tid] = dangling[i] * pr[i];
        __syncthreads();
        for(unsigned int s = 1; s < blockDim.x; s *= 2) {
            int index = 2 * s * tid;
            if (index < blockDim.x) {
                temp[index] += temp[index + s];
            }
        }

        if(tid == 0) atomicAdd(dangling_factor, temp[0]);
    }
}

__global__ void calculateBeta(double *beta, double *dangling_factor, double *alpha, int *V) {

    (*beta) = (*dangling_factor) * (*alpha) / (*V);
}

__global__ void axbp_custom(double *alpha, double *pr_tmp, double *beta, int *personalization_vertex, int *V) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i=thread_id;i<(*V);i+=blockDim.x*gridDim.x){
        pr_tmp[i]=(*alpha) * pr_tmp[i] + (*beta) + ((*personalization_vertex == i) ? (1-(*alpha)) : 0.0);
    }

}

__global__ void euclidean_distance_gpu(double *err,double *pr, double *pr_tmp, int *V) {

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if(thread_id == 0) (*err)=0;
    for(int i=thread_id;i<(*V);i+=blockDim.x*gridDim.x){
        atomicAdd(err, (pr[i]-pr_tmp[i])*(pr[i]-pr_tmp[i]));
    }

}

__global__ void euclidean_distance_gpu_2(double *err, double *pr, double *pr_tmp, int *V) {

    int tid = threadIdx.x;
    int i = tid+blockIdx.x*blockDim.x;

    if(i == 0) (*err)=0;
    if(i<(*V)) {
        __shared__ double temp[DEFAULT_THREADS_PER_BLOCK_VERTEX];
        temp[tid] = (pr[i] - pr_tmp[i]) * (pr[i] - pr_tmp[i]);
        __syncthreads();
        for(unsigned int s = 1; s < blockDim.x; s *= 2) {
            int index = 2 * s * tid;
            if (index < blockDim.x) {
                temp[index] += temp[index + s];
            }
        }

        if(tid == 0) atomicAdd(err, temp[0]);
    }
}

__global__ void axbp_euclidean_distance_gpu(double *alpha, double *pr_tmp, double *beta, int *personalization_vertex, double *err, double *pr, int *V, bool *converged, double convergence_threshold) {
    int tid = threadIdx.x;
    int i = tid+blockIdx.x*blockDim.x;

    if(i == 0) {
        (*err)=0;
        (*converged)=false;
    }
    if(i<(*V)) {
        pr_tmp[i]=(*alpha) * pr_tmp[i] + (*beta) + ((*personalization_vertex == i) ? (1-(*alpha)) : 0.0);
        __shared__ double temp[DEFAULT_THREADS_PER_BLOCK_VERTEX];
        temp[tid] = (pr[i] - pr_tmp[i]) * (pr[i] - pr_tmp[i]);
        __syncthreads();
        for(unsigned int s = 1; s < blockDim.x; s *= 2) {
            int index = 2 * s * tid;
            if (index < blockDim.x) {
                temp[index] += temp[index + s];
            }
        }

        if(tid == 0) atomicAdd(err, temp[0]);
        __syncthreads();
    }
    if(i == 0)
        (*converged) = sqrt(*err) <= convergence_threshold;

}

__device__ double dangling_factor;
__device__ double beta;
__device__ double error;
__device__ bool converged;
__global__ void main_kernel(int *x, int *y, double *val, double *pr, double *pr_tmp, int *dangling, double *alpha, int *personalization_vertex, int max_iterations, double convergence_threshold, int *E, int *V) {

    hipStream_t s1, s2;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);

    //make block size for E
    int threads_per_block = DEFAULT_THREADS_PER_BLOCK;
    int num_blocks = ((*E) + threads_per_block - 1)/ threads_per_block;

    //make block size for V
    int threads_per_block_vertex = DEFAULT_THREADS_PER_BLOCK_VERTEX;
    int num_blocks_vertex = ((*V) + threads_per_block_vertex - 1)/ threads_per_block_vertex;

    int iter = 0;
    converged = false;
    double *temp;

    while (!converged && iter<max_iterations) {

        spmv_coo_gpu<<<num_blocks,threads_per_block,0,s1>>>(x, y, val, pr, pr_tmp, E, V);

        dot_product_gpu_2<<<num_blocks_vertex,threads_per_block_vertex,0,s2>>>(dangling, pr, V, &dangling_factor);

        hipDeviceSynchronize();
        beta = dangling_factor * (*alpha) / (*V);

        axbp_euclidean_distance_gpu<<<num_blocks_vertex,threads_per_block_vertex>>>(alpha, pr_tmp, &beta, personalization_vertex, &error, pr, V, &converged, convergence_threshold);

        hipDeviceSynchronize();
        // Update the PageRank vector;
        temp=pr;
        pr=pr_tmp;
        pr_tmp=temp;

        iter++;
    }
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);
}

// CPU Utility functions;

// Read the input graph and initialize it;
void PersonalizedPageRank::initialize_graph() {
    // Read the graph from an MTX file;
    int num_rows = 0;
    int num_columns = 0;
    read_mtx(graph_file_path.c_str(), &x, &y, &val,
        &num_rows, &num_columns, &E, // Store the number of vertices (row and columns must be the same value), and edges;
        true,                        // If true, read edges TRANSPOSED, i.e. edge (2, 3) is loaded as (3, 2). We set this true as it simplifies the PPR computation;
        false,                       // If true, read the third column of the matrix file. If false, set all values to 1 (this is what you want when reading a graph topology);
        debug,                 
        false,                       // MTX files use indices starting from 1. If for whatever reason your MTX files uses indices that start from 0, set zero_indexed_file=true;
        true                         // If true, sort the edges in (x, y) order. If you have a sorted MTX file, turn this to false to make loading faster;
    );
    if (num_rows != num_columns) {
        if (debug) std::cout << "error, the matrix is not squared, rows=" << num_rows << ", columns=" << num_columns << std::endl;
        exit(-1);
    } else {
        V = num_rows;
    }
    if (debug) std::cout << "loaded graph, |V|=" << V << ", |E|=" << E << std::endl;

    // Compute the dangling vector. A vertex is not dangling if it has at least 1 outgoing edge;
    dangling.resize(V);
    std::fill(dangling.begin(), dangling.end(), 1);  // Initially assume all vertices to be dangling;
    for (int i = 0; i < E; i++) {
        // Ignore self-loops, a vertex is still dangling if it has only self-loops;
        if (x[i] != y[i]) dangling[y[i]] = 0;
    }
    // Initialize the CPU PageRank vector;
    pr.resize(V);
    pr_golden.resize(V);
    // Initialize the value vector of the graph (1 / outdegree of each vertex).
    // Count how many edges start in each vertex (here, the source vertex is y as the matrix is transposed);
    int *outdegree = (int *) calloc(V, sizeof(int));
    for (int i = 0; i < E; i++) {
        outdegree[y[i]]++;   /////this is done in initKernelOutdegree
        //printf("%d",outdegree[y[i]]);
    }
    // Divide each edge value by the outdegree of the source vertex;
    for (int i = 0; i < E; i++) {
    //// each node val is dependent on the number of outgoing edges
        val[i] = 1.0 / outdegree[y[i]];  /////this is done in initKernelInverse
    }
    free(outdegree);
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void PersonalizedPageRank::alloc() {
    // Load the input graph and preprocess it;
    initialize_graph();

    // Size of allocations
    V_size = V * sizeof(double);
    dangling_size = V * sizeof(int);
    E_size = E * sizeof(int);
    val_size = E * sizeof(double);



    // Allocate space in VRAM
    hipMalloc((void **)&x_gpu, E_size);
    hipMalloc((void **)&y_gpu, E_size);
    hipMalloc((void **)&val_gpu, val_size);
    hipMalloc((void **)&pr_gpu, V_size);
    hipMalloc((void **)&pr_tmp_gpu, V_size);
    hipMalloc((void **)&alpha_gpu, sizeof(double));
    hipMalloc((void **)&V_gpu, sizeof(int));
    hipMalloc((void **)&E_gpu, sizeof(int));
    hipMalloc((void **)&personalization_vertex_gpu, sizeof(int));

    if(implementation != 1) {
        hipMalloc((void **)&dangling_gpu, dangling_size);
    } else {
        hipMalloc((void **)&dangling_factor_gpu, sizeof(double));
        hipMalloc((void **)&beta_gpu, sizeof(double));
        hipMalloc((void **)&error_gpu, sizeof(double));
        hipMalloc((void **)&temp_gpu, sizeof(double));
        hipMalloc((void **)&dangling_double_gpu, V_size);
        hipMalloc((void **)&converged_gpu, sizeof(bool));
    }

}

// Initialize data;
void PersonalizedPageRank::init() {
    // Do any additional CPU or GPU setup here;

    hipMemcpy(x_gpu, &x[0], E_size, hipMemcpyHostToDevice);
    hipMemcpy(y_gpu, &y[0], E_size, hipMemcpyHostToDevice);
    hipMemcpy(val_gpu, &val[0], val_size, hipMemcpyHostToDevice);
    hipMemcpy(alpha_gpu, &alpha, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(V_gpu, &V, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(E_gpu, &E, sizeof(int), hipMemcpyHostToDevice);

    if(implementation != 1) {
        hipMemcpy(dangling_gpu, &dangling[0], dangling_size, hipMemcpyHostToDevice);
    } else {
        std::vector<double> dangling_tmp(dangling.begin(), dangling.end());
        hipMemcpy(dangling_double_gpu, &dangling_tmp[0], V * sizeof(double), hipMemcpyHostToDevice);
    }
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void PersonalizedPageRank::reset() {
    // Reset the PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr.begin(), pr.end(), 1.0 / V);
    hipMemcpy(pr_gpu, &pr[0], V_size, hipMemcpyHostToDevice);

    // Generate a new personalization vertex for this iteration;
    personalization_vertex = rand() % V;
    if (debug) std::cout << "personalization vertex=" << personalization_vertex << std::endl;

    // Do any GPU reset here, and also transfer data to the GPU;
    hipMemcpy(personalization_vertex_gpu, &personalization_vertex, sizeof(int), hipMemcpyHostToDevice);
}

void PersonalizedPageRank::execute(int iteration) {
    // Do the GPU computation here, and also transfer results to the CPU;

    if(implementation != 1) {
        main_kernel<<<1,1>>>(x_gpu, y_gpu, val_gpu, pr_gpu, pr_tmp_gpu, dangling_gpu, alpha_gpu, personalization_vertex_gpu, max_iterations, convergence_threshold, E_gpu, V_gpu);

    } else {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

        //make block size for E
        int threads_per_block = DEFAULT_THREADS_PER_BLOCK;
        int num_blocks = (E + threads_per_block - 1)/ threads_per_block;

        //make block size for V
        int threads_per_block_vertex = DEFAULT_THREADS_PER_BLOCK_VERTEX;
        int num_blocks_vertex = (V + threads_per_block_vertex - 1)/ threads_per_block_vertex;

        int iter = 0;
        bool converged = false;

        while (!converged && iter<max_iterations) {

            spmv_coo_gpu<<<num_blocks,threads_per_block>>>(x_gpu, y_gpu, val_gpu, pr_gpu, pr_tmp_gpu, E_gpu, V_gpu);

            hipblasDdot(handle, V, dangling_double_gpu, 1, pr_gpu, 1, dangling_factor_gpu);

            calculateBeta<<<1,1>>>(beta_gpu, dangling_factor_gpu, alpha_gpu, V_gpu);

            axbp_euclidean_distance_gpu<<<num_blocks_vertex,threads_per_block_vertex>>>(alpha_gpu, pr_tmp_gpu, beta_gpu, personalization_vertex_gpu, error_gpu, pr_gpu, V_gpu, converged_gpu, convergence_threshold);

            hipMemcpy(&converged, converged_gpu, sizeof(bool), hipMemcpyDeviceToHost);

            // Update the PageRank vector;
            temp_gpu=pr_gpu;
            pr_gpu=pr_tmp_gpu;
            pr_tmp_gpu=temp_gpu;

            iter++;
        }
        hipblasDestroy(handle);
    }

    hipMemcpy(&pr[0], pr_gpu, V_size, hipMemcpyDeviceToHost);
}

void PersonalizedPageRank::cpu_validation(int iter) {

    // Reset the CPU PageRank vector (uniform initialization, 1 / V for each vertex);
    std::fill(pr_golden.begin(), pr_golden.end(), 1.0 / V);

    // Do Personalized PageRank on CPU;
    auto start_tmp = clock_type::now();
    personalized_pagerank_cpu(x.data(), y.data(), val.data(), V, E, pr_golden.data(), dangling.data(), personalization_vertex, alpha, convergence_threshold, max_iterations);
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    if(debug) std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;

    // Obtain the vertices with highest PPR value;
    std::vector<std::pair<int, double>> sorted_pr_tuples = sort_pr(pr.data(), V);
    std::vector<std::pair<int, double>> sorted_pr_golden_tuples = sort_pr(pr_golden.data(), V);

    // Check how many of the correct top-20 PPR vertices are retrieved by the GPU;
    std::set<int> top_pr_indices;
    std::set<int> top_pr_golden_indices;
    int old_precision = std::cout.precision();
    std::cout.precision(4);
    int topk = std::min(V, topk_vertices);
    for (int i = 0; i < topk; i++) {
        int pr_id_gpu = sorted_pr_tuples[i].first;
        int pr_id_cpu = sorted_pr_golden_tuples[i].first;
        top_pr_indices.insert(pr_id_gpu);
        top_pr_golden_indices.insert(pr_id_cpu);
        if (debug) {
            double pr_val_gpu = sorted_pr_tuples[i].second;
            double pr_val_cpu = sorted_pr_golden_tuples[i].second;
            if (pr_id_gpu != pr_id_cpu) {
                std::cout << "* error in rank! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            } else if (std::abs(sorted_pr_tuples[i].second - sorted_pr_golden_tuples[i].second) > convergence_threshold) {
                std::cout << "* error in value! (" << i << ") correct=" << pr_id_cpu << " (val=" << pr_val_cpu << "), found=" << pr_id_gpu << " (val=" << pr_val_gpu << ")" << std::endl;
            }
        }
    }
    precision_to_print = precision;
    std::cout.precision(old_precision);
    // Set intersection to find correctly retrieved vertices;
    std::vector<int> correctly_retrieved_vertices;
    set_intersection(top_pr_indices.begin(), top_pr_indices.end(), top_pr_golden_indices.begin(), top_pr_golden_indices.end(), std::back_inserter(correctly_retrieved_vertices));
    precision = double(correctly_retrieved_vertices.size()) / topk;
    if (debug) std::cout << "correctly retrived top-" << topk << " vertices=" << correctly_retrieved_vertices.size() << " (" << 100 * precision << "%)" << std::endl;
}

std::string PersonalizedPageRank::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(precision_to_print);
    } else {
        // Print the first few PageRank values (not sorted);
        std::ostringstream out;
        out.precision(3);
        out << "[";
        for (int i = 0; i < std::min(20, V); i++) {
            out << pr[i] << ", ";
        }
        out << "...]";
        return out.str();
    }
}


void PersonalizedPageRank::clean() {
    // Delete any GPU data or additional CPU data;
    hipFree(x_gpu);
    hipFree(y_gpu);
    hipFree(val_gpu);
    hipFree(pr_gpu);
    hipFree(pr_tmp_gpu);
    hipFree(V_gpu);
    hipFree(alpha_gpu);
    hipFree(personalization_vertex_gpu);
    hipFree(E_gpu);

    if(implementation != 1) {
        hipFree(dangling_gpu);
    } else {
        hipFree(dangling_factor_gpu);
        hipFree(beta_gpu);
        hipFree(error_gpu);
        hipFree(temp_gpu);
        hipFree(dangling_double_gpu);
        hipFree(converged_gpu);
    }
}
